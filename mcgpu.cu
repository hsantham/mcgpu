#include "hip/hip_runtime.h"
//extern "C"

#include<stdio.h>
#include <sys/stat.h>
#include <assert.h>
#include <stdint.h>

#define TERMS_PER_CHUNK 1000

// TYPE DEFs
typedef char * FileName;
typedef unsigned char * FileContent;
typedef unsigned long long int BigBoy;

// User defined data types
struct chunkOrder {
  BigBoy chunkOffset;
  BigBoy reorderedPosition;
};

struct termvector {
    unsigned char *term;
    unsigned short count;
};

struct scores {
    double score;
    unsigned int index;
};

typedef struct scores SCORE;

typedef struct termvector TermVector;

__device__ __host__   BigBoy findNC2(int n) {
return ((n * (n-1))/2);
}

// USER DEFINED TYPE DEFs
typedef struct chunkOrder ChunkOrder;

// Global variables
FileName filename, outputFilename;

__device__ __host__  void preprocess(FileContent   fileContent, unsigned int myIdx, unsigned int myLimit) {
    for(unsigned int i = myIdx; i < myLimit; i++) {
        switch(fileContent[i]) {
            case '.':
            case '(':
            case ')':
            case ':':
            case '-':
            case ',':
                fileContent[i] = ' ';
                break;
        }
    }
}

__host__ __device__ unsigned int isMatch(unsigned char *term1, unsigned char *term2) {
    while(*term1 == *term2) {
        if(*term1 == '\0' || *term1 == ' ') return 1;
        term1++; term2++;
    }
    
    return 0;
}


void printVector(TermVector *vector, unsigned int used, unsigned char *db, unsigned char *hb) {
    unsigned int i;
    for(i = 0; i < used; i++) {
        unsigned int j=0;
        unsigned char *term = hb + (vector[i].term - db);
        while(term[j] != ' ' && term[j] != '\0') {
            printf("%c", term[j]);
            j++;
        }
        printf("   = %d\n",vector[i].count);
    }
}

__host__ __device__ unsigned int findIndex(TermVector *vector1, unsigned int vector1Count, FileContent term) {
    unsigned int j;
    for(j = 0; j < vector1Count; j++) {
        if(isMatch(vector1[j].term, term)) {
            return j;
        }
    }

    return (unsigned int) -1;
}

__host__ __device__ unsigned int findLength(FileContent   content) {
        unsigned int i = 0;
        while(content[i] != ' ' && content[i] != '\0')
                i++;
        return i;
}

#define VECTOR_SIZE 1300
__host__ __device__ void getVector(FileContent   content1, 
                unsigned int  letterCount1,
                TermVector *vector1,
                unsigned int *used) {
    unsigned int i = 0, startIdx = (unsigned int)-1;
    for(; i < letterCount1; i++) {
        if(content1[i] != ' ') {
            startIdx = (startIdx == (unsigned int)-1) ? i : startIdx;
            continue;
        }

        if(startIdx == (unsigned int)-1) {
            continue;
        }


        unsigned int index = findIndex(vector1, *used, content1+startIdx);
        if(index == (unsigned int)-1) {
            assert(*used < VECTOR_SIZE);
            if(i - startIdx > 1) {
                vector1[*used].term = content1+startIdx;
                vector1[*used].count = 1;
                (*used)++;
            }
        } else {
            vector1[index].count++;
        }
        startIdx = (unsigned int)-1;
    }

    if(startIdx != (unsigned int)-1) {
        unsigned int index = findIndex(vector1, *used, content1+startIdx);
        if(index == (unsigned int)-1) {
            if(findLength(content1+startIdx) >= 3) {
                assert(*used < VECTOR_SIZE);
                vector1[*used].term = content1+startIdx;
                vector1[*used].count = 1;
                (*used)++;
            }
        } else {
            vector1[index].count++;
        }
        
    }
}

__host__ __device__ unsigned int getADotB(TermVector *vector1, 
                      unsigned int vector1Count,
                      TermVector *vector2,
                      unsigned int vector2Count) {
    unsigned int i;
    unsigned int aDotb = 0;
    for(i = 0; i < vector1Count; i++) {
        unsigned int index = findIndex(vector2, vector2Count, vector1[i].term);
        if(index != (unsigned int)-1) {
            aDotb += (vector1[i].count * vector2[index].count);
        }
    }

    return aDotb;
}

__host__ __device__ double modOfVector(TermVector *vector, unsigned int vectorCount) {
    unsigned int i;
    unsigned int modValue = 0;
    for(i = 0; i < vectorCount; i++) {
        modValue += (vector[i].count * vector[i].count);
    }

    return sqrt((double) modValue);
}

__device__ __host__  double getScore(TermVector *vector1, 
                           unsigned int vector1Count,
                           TermVector *vector2,
                           unsigned int vector2Count) {
    unsigned int dotProduct = getADotB(vector1, vector1Count, vector2, vector2Count); 
    double magA = modOfVector(vector1, vector1Count);
    double magB = modOfVector(vector2, vector2Count);
    return (double)dotProduct/(magA * magB);
}

__device__ __host__ void sqWrapper(FileContent   fileContent, 
                   BigBoy        fileSize, 
                   BigBoy        noOfChunks,
                   BigBoy        chunkSize,
                   SCORE*        dScores,
                   unsigned int* dSyncBuffer,
                   TermVector*   dTermVector,
                   unsigned int* dUsedArr,
                   unsigned int preprocessing,
				   unsigned int myIdx)
{
    if(preprocessing)
        preprocess(fileContent, myIdx * chunkSize, min(fileSize, (myIdx+1) * chunkSize));

#ifndef SERIAL
	//__syncthreads();
#endif

    unsigned int used = 0;
    TermVector *myVector = dTermVector + myIdx * TERMS_PER_CHUNK;
    getVector(fileContent + myIdx * chunkSize, 
              min(chunkSize, fileSize - myIdx * chunkSize),
              myVector,
              &used);
    dUsedArr[myIdx] = used;

    if(myIdx == 0) 
        memset(dScores, 0, sizeof(SCORE)*noOfChunks*noOfChunks);

}

__global__ void sq(FileContent   fileContent, 
                   BigBoy        fileSize, 
                   BigBoy        noOfChunks,
                   BigBoy        chunkSize,
                   SCORE*        dScores,
                   unsigned int* dSyncBuffer,
                   TermVector*   dTermVector,
                   unsigned int* dUsedArr,
                   unsigned int preprocessing)
{
    unsigned int myIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if(noOfChunks <= myIdx) return;

	sqWrapper(fileContent, fileSize, noOfChunks, chunkSize, 
	dScores, dSyncBuffer, dTermVector, dUsedArr, preprocessing, myIdx);
}

__device__ __host__ double getScoreWrapper(TermVector* dTermVector, 
										unsigned int *dUsedArr,
										BigBoy noOfChunks, 
										SCORE* dScores, 
										unsigned int firstChunk, 
										unsigned int secondChunk) {
    double score = getScore(dTermVector + firstChunk * TERMS_PER_CHUNK,
                            dUsedArr[firstChunk],
                            dTermVector + secondChunk * TERMS_PER_CHUNK,
                            dUsedArr[secondChunk]);
    dScores[firstChunk * noOfChunks + secondChunk].score = score;
    dScores[firstChunk * noOfChunks + secondChunk].index = secondChunk;
    dScores[secondChunk * noOfChunks + firstChunk].score = score;
    dScores[secondChunk * noOfChunks + firstChunk].index = firstChunk;
	return score;
}

__global__ void computeScore(TermVector*   dTermVector,
                             unsigned int* dUsedArr,
                             BigBoy        noOfChunks,
                             SCORE*        dScores) {
    unsigned int myIdx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int nc2 = findNC2(noOfChunks);
    if(nc2 <= myIdx) return;

    unsigned int firstChunk, secondChunk, i, j, k = 0; 
    for(i = 0; i <= (noOfChunks-1-1); i++) {
        for(j = i+1; j <= (noOfChunks-1); j++) {
            if(myIdx == k) {
                firstChunk = i;
                secondChunk = j;
                i = noOfChunks; // To break from outer for loop
                break;
            }
            k++;
        }
    } 

	getScoreWrapper(dTermVector,
                    dUsedArr,
                    noOfChunks,
                    dScores,
	                firstChunk,
					secondChunk);
}

__device__ __host__ void iSort(SCORE *arr, unsigned int n) {
    int i, j;
    for(i = 1; i < n; i++) {
        SCORE tmp = arr[i];
        for(j = i - 1; j >= 0; j--) {
            if(arr[j].score > tmp.score) {
                break;
            }
            arr[j+1] = arr[j];
        }
        arr[j+1] = tmp;
    }
}

#define sumOfN(n) ((n) *((n)+1))/2

__global__ void sortScores(SCORE*       dScores,
                           BigBoy        noOfChunks) {
    unsigned int myIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if(myIdx >= noOfChunks) return;

    iSort(dScores + myIdx * noOfChunks, noOfChunks);
}
                              
__host__ __device__ unsigned int findIndex(SCORE*       dScores,
                                  unsigned int n,
                                  BigBoy        noOfChunks,
                                  unsigned int kthBest) {
    return dScores[n*noOfChunks + kthBest].index;
}

__device__ __host__ unsigned int isAlreadyThere(unsigned int *order,
                                       unsigned int n,
                                       unsigned int index) {
    for(unsigned int i = 0; i<n; i++) {
        if(order[i] == index) return 1;
    }

    return 0;
}

__device__ __host__ int isIndex32kApart(unsigned int index1, unsigned int index2, BigBoy chunksize, unsigned int distThreshold) {
        unsigned int WSIZE = distThreshold - 258 - 3;
       if(index1 > index2) 
               return ((index1 - index2) *chunksize) >= WSIZE;

       return ((index2 - index1) *chunksize) >= WSIZE;
}

#define dabs(a,b) ((a)>(b)?(a-b):(b-a))

__device__ __host__ int findUnusedNearestIndex(unsigned int *order, unsigned int soFarFilled, unsigned int index, unsigned int noOfChunks) {
        unsigned int min = (unsigned int) -1;
        for(unsigned int i=0; i < noOfChunks; i++) {
                if(i == index) continue;
                
                unsigned int continuee = 0;
                for(unsigned int k = 0; k < soFarFilled; k++) {
                        if(order[k] == i) {continuee=1; break;}
                }
                if(continuee == 1) continue;

                if(min == (unsigned int)-1) 
                {
                        min = i;
                }

                if(dabs(min, index) > dabs(i, index))
                {
                        min = i;        
                }
        }

        return min;
}


__device__ __host__ double findScore(SCORE*       dScores, BigBoy        noOfChunks, unsigned int index) {

for(int i=0; i<noOfChunks; i++) {
    if(dScores[i].index == index) return dScores[i].score;
}
assert(0);

return 0.0;
}
__host__ __device__ void getOrderWrapper(SCORE*       dScores,
                         BigBoy        noOfChunks,
                         BigBoy        chunksize,
                         float  threshold,
                         unsigned int distThreshold) {
    unsigned int *order = (unsigned int *) malloc(sizeof(unsigned int) * noOfChunks);
    unsigned int i, k;
    order[0] = 0;

    for(i=1; i<noOfChunks; i++) {
        order[i] = i;
        k=0;
        while(1) {
            unsigned int index = findIndex(dScores, order[i-1], noOfChunks, k);
            if((findScore(dScores + order[i-1] * noOfChunks, noOfChunks, index) > (double)threshold) &&
               !isAlreadyThere(order, i, index) && isIndex32kApart(order[i-1], index, chunksize, distThreshold)) {
                order[i] = index;
                break;
            }
            k++;

            if(k == noOfChunks) {
                for(int j=0; j < noOfChunks; j++) {
                    if(!isAlreadyThere(order, i, j)) {
                        order[i] = j;
                        break;
                    }
                }
                break;
            }
        }
    }

    for(i = 0; i < noOfChunks; i++) {
        dScores[i].score = order[i];
        dScores[i].index = order[i];
    }
}
	
__global__ void getOrder(SCORE*       dScores,
                         BigBoy        noOfChunks,
                         BigBoy        chunksize,
                         float  threshold,
                         unsigned int distThreshold) {
    getOrderWrapper(dScores, noOfChunks, chunksize, threshold, distThreshold);
}

size_t getFilesize(FileName filename) {
    struct stat st;
    if(stat((const char *)filename, &st) != 0) {
        assert(0);
        return 0;
    }
    return st.st_size;   
}

void getFileContent(FileName filename, FileContent buffer) {
    FILE *file;
    size_t nread;

    file = fopen((const char *)filename, "r");
    assert(file != NULL);

    BigBoy filesize = getFilesize(filename);

    nread = fread(buffer, 1, filesize, file);
    assert(nread == filesize);
    assert(ferror(file) == 0);

    fclose(file);
}

void getOrderFromFile(SCORE *hScores) {
    FILE *file;
    size_t nread;

    char *fn = (char *) "order";
    file = fopen((const char *)fn, "r");
    assert(file != NULL);

    BigBoy filesize = getFilesize(fn);

    char *buffer = (char *) malloc(filesize+1);
    assert(buffer != NULL);
    nread = fread(buffer, 1, filesize, file);
    assert(nread == filesize);
    assert(ferror(file) == 0);

    int startIdx = 0, k = 0;
    for(int i=0; i<nread; i++) {
        if(buffer[i] == ' ') {
            buffer[i] = '\0';
            hScores[k++].index = atoi(buffer+startIdx);
            startIdx = i + 1;           
        } 
    }
    hScores[k++].index = atoi(buffer+startIdx);

    fclose(file);
}

#ifndef checkCudaErrors
#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)

// These are the inline versions for all of the SDK helper functions
inline void __checkCudaErrors(int err, const char *file, const int line)
{
    if (0 != err)
    {
        fprintf(stderr, "checkCudaErrors() Driver API error = %04d  from file <%s>, line %i.\n",
                err, file, line);
        exit(EXIT_FAILURE);
    }
}
#endif

void preprocessFileForSpace(char *inputFile, char *outputFile, unsigned int chunkSize) {
    FILE *file;
    size_t nread;

    file = fopen((const char *)inputFile, "r");
    assert(file != NULL);

    BigBoy filesize = getFilesize(inputFile);
    char *buffer = (char *) malloc(filesize+1);
    char *temp = buffer;
    assert(buffer != NULL);
    
    nread = fread(buffer, 1, filesize, file);
    assert(nread == filesize);
    assert(ferror(file) == 0);

    fclose(file);

    file = fopen((const char *)outputFile, "w");
    assert(file != NULL);

    BigBoy noOfChunks = (int) ceil(((double)filesize/(double)chunkSize));
    size_t written;
    for(unsigned int i = 0; i < noOfChunks; i++) {
        unsigned int bytesToWrite = min(chunkSize, (unsigned int) filesize - i * chunkSize);
        buffer[bytesToWrite - 1] = ' ';
        written = fwrite(buffer, 1, bytesToWrite, file);
        assert(written == bytesToWrite);
        assert(ferror(file) == 0);

        buffer += written;
    }

    fclose(file);

    free(temp);
}

unsigned int getNoOfDigits(unsigned int index) {
    unsigned int reverse = 1;
    while(index != 0) {
        index = index / 10;
        if(index) 
            reverse++;
    }

    return reverse;
}

unsigned int getReverse(unsigned int index) {
    unsigned int reverse = 0;
    while(index != 0) {
        reverse = reverse * 10 + (index % 10);
        index = index / 10;
    }

    return reverse;
}

#define PRINT_DEBUG 1
int main(int argc, char *argv[]) {

    unsigned int i, j;
    filename = (char *) "input_text";
    FileName pfilename = ( char *) "ptext8";

    BigBoy chunkSize = (argc < 2) ? 1024 : atoi(argv[1]);
    float threshold = (argc < 3) ? 0.0 : atof(argv[2]);
    int doPreprocess = (argc < 4) ? 0 : atoi(argv[3]);
    unsigned int manualOrder = (argc < 5) ? 0 : atoi(argv[4]);
    unsigned int printVectorArg = (argc < 6) ? 0 : atoi(argv[5]);
    unsigned int distThreshold = (argc < 7) ? 0x8000 : (atoi(argv[6])+258+3);

    printf("Input args:: Chunk-size:  %u; Threshold:   %f; Preprocessing-ptext8: %d;\n"
           "             ManualOrder: %d; printVector: %d; distThreshold:        %d; \n\n", 
            chunkSize, threshold, doPreprocess, manualOrder, printVectorArg, distThreshold);

    if(doPreprocess)
    {
        printf("0) Pre-processing ptext -> text8\n");
        preprocessFileForSpace(pfilename, filename, chunkSize);
    }
    FileContent deviceFileBuffer, hostFileBuffer;
    BigBoy filesize = getFilesize(filename);

    BigBoy noOfChunks = (int) ceil(((double)filesize/(double)chunkSize));
    BigBoy nc2 = findNC2(noOfChunks);
    unsigned int threadsPerBlock = 16;
    unsigned int noOfBlocks = (int) ceil(((double)nc2/(double)threadsPerBlock));

    printf("File size = %llu; chunk size = %llu;    no of chunks = %llu; nc2 = %llu\n", filesize, chunkSize, noOfChunks, nc2);

    TermVector *hVector;
    unsigned int *hUsedArr;
    SCORE *hScores;
    hostFileBuffer = (FileContent) malloc(filesize+1);
    getFileContent(filename, hostFileBuffer);

    hScores = (SCORE *) malloc(sizeof(SCORE)*noOfChunks*noOfChunks);
    assert(hScores != NULL);
    hVector = (TermVector*) malloc(noOfChunks * TERMS_PER_CHUNK * sizeof(TermVector));
    assert(hVector != NULL);
    hUsedArr = (unsigned int *) malloc(noOfChunks * sizeof(unsigned int));
    assert(hUsedArr != NULL);

    printf("0) Computing vectors for n chunks\n\n");
#ifndef SERIAL
    SCORE *dScores;
    unsigned int *dSyncBuffer, *dUsedArr;
    TermVector *dVector;
    checkCudaErrors(hipMalloc((void **)&deviceFileBuffer, filesize));
    checkCudaErrors(hipMalloc((void **)&dScores, sizeof(SCORE)*noOfChunks*noOfChunks));
    checkCudaErrors(hipMalloc((void **)&dSyncBuffer, sizeof(unsigned int)*10));
    checkCudaErrors(hipMalloc((void **)&dVector, noOfChunks * TERMS_PER_CHUNK * sizeof(TermVector)));
    checkCudaErrors(hipMalloc((void **)&dUsedArr, noOfChunks * sizeof(unsigned int)));

    checkCudaErrors(hipMemcpy(deviceFileBuffer, hostFileBuffer, filesize, hipMemcpyHostToDevice));

    threadsPerBlock = 1;
    noOfBlocks = ceil((double)noOfChunks/(double)threadsPerBlock);
    sq<<<noOfBlocks, threadsPerBlock>>>(deviceFileBuffer, 
                                        filesize, 
                                        noOfChunks, 
                                        chunkSize, 
                                        dScores, 
                                        dSyncBuffer, 
                                        dVector,
                                        dUsedArr, doPreprocess);
   
	if(printVectorArg) {
        checkCudaErrors(hipMemcpy(hVector, dVector, noOfChunks * TERMS_PER_CHUNK * sizeof(TermVector), hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(hUsedArr, dUsedArr, noOfChunks * sizeof(unsigned int), hipMemcpyDeviceToHost));
    }
#else
	for(int i=0;i<noOfChunks;i++) {
		sqWrapper(hostFileBuffer, 
				filesize, 
				noOfChunks, 
				chunkSize, 
				hScores, 
				NULL, 
				hVector,
				hUsedArr, doPreprocess, i);
	}
#endif
	
	if(printVectorArg) {
        for(int i = 0; i< noOfChunks; i++) {
            printf("Printing chunk %d::\n",i);
            printVector(hVector + i * TERMS_PER_CHUNK , hUsedArr[i], hostFileBuffer, hostFileBuffer); 
        }
    }

    

    printf("1) Comppute score started\n\n");
#ifndef SERIAL
    noOfBlocks = ceil((double)nc2/(double)256);;
    threadsPerBlock = 256;
    computeScore<<<noOfBlocks, threadsPerBlock>>>(dVector, dUsedArr, noOfChunks, dScores);
    checkCudaErrors(hipMemcpy(hScores, 
                               dScores, 
                               sizeof(SCORE)*noOfChunks*noOfChunks, 
                               hipMemcpyDeviceToHost));
#else
	for(int fc=0; fc < (noOfChunks-1); fc++) {
		for(int sc=fc+1;sc < noOfChunks; sc++) {
			getScoreWrapper(hVector, hUsedArr, noOfChunks, hScores, fc, sc);
		}
	}
#endif

#ifdef PRINT_DEBUG
    if(noOfChunks <= 10) {
    for(i = 0; i < noOfChunks; i++) {
        for(j = 0; j < noOfChunks; j++) {
            printf("%lf,%u",hScores[i*noOfChunks + j].score, hScores[i*noOfChunks + j].index);
            printf(" ");
        }
        printf("\n");
    }
    }
#endif

    printf("2) Sort score started\n\n");

#ifndef SERIAL
    threadsPerBlock = 1;
    noOfBlocks = ceil((double)noOfChunks/(double)threadsPerBlock);
	
    //sortScores<<<noOfBlocks, threadsPerBlock>>>(dScores, noOfChunks);
    checkCudaErrors(hipMemcpy(hScores, 
                               dScores, 
                               sizeof(SCORE)*noOfChunks*noOfChunks, 
                               hipMemcpyDeviceToHost));
	for(i = 0; i < noOfChunks; i++) {
		iSort(hScores + i * noOfChunks, noOfChunks);
	}
#else
	for(i = 0; i < noOfChunks; i++) {
		iSort(hScores + i * noOfChunks, noOfChunks);
	}
#endif

#ifdef PRINT_DEBUG
    if(noOfChunks <= 10) {
    for(i = 0; i < noOfChunks; i++) {
        for(j = 0; j < noOfChunks; j++) {
            printf("%lf,%u",hScores[i*noOfChunks + j].score, hScores[i*noOfChunks + j].index);
            printf(" ");
        }
        printf("\n");
    }
    }
#endif

    if(manualOrder) {
        printf("3) Getting order from file instead of actual computation !!! \n\n");
        getOrderFromFile(hScores);
    }
    else {
        printf("3) Getting order from actual data\n\n");
#ifndef SERIAL
        //getOrder<<<1,1>>>(dScores, noOfChunks, chunkSize, threshold, distThreshold);
        //checkCudaErrors(hipMemcpy(hScores, 
        //                          dScores, 
        //                           sizeof(SCORE)*noOfChunks*noOfChunks, 
        //                           hipMemcpyDeviceToHost));
		getOrderWrapper(hScores, noOfChunks, chunkSize, threshold, distThreshold);
#else
		getOrderWrapper(hScores, noOfChunks, chunkSize, threshold, distThreshold);
#endif
    }

    for(i = 0; i < noOfChunks; i++) {
        for(j = 0; j < noOfChunks; j++) {
            printf("%u ", hScores[i*noOfChunks + j].index);
        }
        printf("\n");
        break;
    }
    
    FILE *file;
    outputFilename = (char *) "reorder_info";
    file = fopen((const char *)outputFilename, "w");
    assert(file != NULL);

    size_t written;
    for(i = 0; i <= noOfChunks; i++) {
        unsigned int index;
        if(i != noOfChunks) {
            index = hScores[i].index; 
        } else {
            index = chunkSize;
        }
        
        unsigned int reversed = getReverse(index);
        unsigned int noOfDigits = getNoOfDigits(index);
        for(int k=0; k<noOfDigits; k++){
            char b='0' + (reversed%10);
            written = fwrite(&b, 1, 1, file);
            assert(written == 1);
            assert(ferror(file) == 0);
            reversed = reversed/10;
        }
        if(i != (noOfChunks)) {
            char b=' ';
            written = fwrite(&b, 1, 1, file);
            assert(written == 1);
            assert(ferror(file) == 0);
        }
    }

    fclose(file);

#ifndef SERIAL
    checkCudaErrors(hipFree(deviceFileBuffer));
    checkCudaErrors(hipFree(dScores));
    checkCudaErrors(hipFree(dSyncBuffer));
    checkCudaErrors(hipFree(dVector));
    checkCudaErrors(hipFree(dUsedArr));
#endif

    // open file descriptor
    outputFilename = (char *) "preprocessed_input_text";
    file = fopen((const char *)outputFilename, "w");
    assert(file != NULL);

    for(i = 0; i < noOfChunks; i++) {
        unsigned int bytesToWrite = min(chunkSize, filesize - hScores[i].index * chunkSize);
        written = fwrite(hostFileBuffer + hScores[i].index * chunkSize, 1, bytesToWrite, file);
        assert(written == bytesToWrite);
        assert(ferror(file) == 0);
    }

    fclose(file);

    return 0;
}
