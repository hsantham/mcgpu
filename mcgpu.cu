#include "hip/hip_runtime.h"
extern "C"
{
#include<stdio.h>
#include <sys/stat.h>
#include <assert.h>
#include <stdint.h>
}

// TYPE DEFs
typedef unsigned char * FileName;
typedef unsigned char * FileContent;
typedef unsigned long long int BigBoy;

// User defined data types
struct chunkOrder {
  BigBoy chunkOffset;
  BigBoy reorderedPosition;
};

struct llnode {
    void *data;
    struct llnode *next;
};

typedef struct llnode LLNode;

struct hnode {
    unsigned char *string;
    unsigned char source;
    unsigned char count;
};
typedef struct hnode HNode;

__device__ LLNode *getNewLLNode(void *data) {
    LLNode *head = (LLNode *) malloc(sizeof(LLNode));
    head->data = data;
    head->next = NULL;

    return head;
}

__device__ void append(LLNode *head, LLNode *nodeToInsert) {
  LLNode *node = head;

  while(node->next) node=node->next;

  node->next = nodeToInsert;
  nodeToInsert->next = NULL;
}

extern "C"
__device__ __host__ BigBoy findNC2(int n) {
return ((n * (n-1))/2);
}

// USER DEFINED TYPE DEFs
typedef struct chunkOrder ChunkOrder;

// Global variables
FileName filename;

extern "C"
__device__ void preprocess(FileContent   fileContent, int myIdx, int myLimit) {
    for(int i = myIdx; i < myLimit; i++) {
        switch(fileContent[i]) {
            case '.':
            case '(':
            case ')':
            case ':':
            case '-':
            case ',':
                fileContent[myIdx] = ' ';
                break;
        }
    }
}

#define MAX_HASH_TABLE_ENTRIES 20
#define MAX_NODES_IN_HASH_TABLE 250

__device__ uint32_t getHash(FileContent string, uint32_t length) {
    uint32_t hash = 0;
    for(uint32_t i = 0; i < length; i++) {
        hash = hash + string[i];
    }

    return hash % MAX_HASH_TABLE_ENTRIES;
}

__device__ HNode* getHashNode(FileContent string, 
                   uint8_t     source, 
                   uint8_t     count, 
                   HNode*      nodes, 
                   uint32_t*   used) {
    HNode* node = nodes + *used;
    node->string = string;
    node->source = source;
    node->count  = count;

    *(used)++;
    return node;
} 

__device__ void fillHashTable(LLNode **hashTable, uint32_t hash, LLNode *node) {
    LLNode *head = hashTable[hash];
    if(head == NULL) {
        hashTable[hash] = head;
    } else {
        while(head->next) head = head->next;
        head->next = node;
    }
}

__device__ LLNode* findMatchingNode(LLNode* head, FileContent content) {
    LLNode *node = head;
    while(node) {
        FileContent existing = (FileContent) node->data;
        for(uint64_t i = 0;; i++) {
            if(existing[i] == content[i]) {
                continue;
            }
            
            if(existing[i] == ' ' && content[i] == ' ') {
                return node;
            }

            if(existing[i] == '\0' && content[i] == '\0') {
                return node;
            }

            break;
        }
    }

    return NULL;
}

extern "C"
__device__ double getScoreFromHashTable(LLNode **hashTable) {
    double euclidDist = 0;
    for(uint32_t i = 0; i < MAX_HASH_TABLE_ENTRIES; i++) {
        LLNode *node = hashTable[i];
        while(node) {
            HNode *hnode = (HNode *) node->data;            
            euclidDist += hnode->count * hnode->count;
            node = node->next;
        }
    }

    return sqrt((double) euclidDist);
}

extern "C"
__device__ double getScore(FileContent   content1, 
                           unsigned int  letterCount1, 
                           FileContent   content2, 
                           unsigned int  letterCount2) {
    LLNode *hashTable[MAX_HASH_TABLE_ENTRIES] = {};
    
    int i = 0, startIdx=-1;
    HNode nodes[MAX_NODES_IN_HASH_TABLE]; uint32_t used = 0;
    unsigned char currentSrc = 1;
    for(;i<letterCount1;i++) {
        if(content1[i] != ' ') {
            startIdx = (startIdx == -1) ? i : startIdx;
            continue;
        }

        if(startIdx == -1) {
            continue;
        }

        unsigned char hash = getHash(content1+startIdx, i-startIdx);
        LLNode *head = hashTable[hash];
        
        if(head == NULL) {
            HNode *hashNode = getHashNode(content1+startIdx, currentSrc, 1, nodes, &used);
            LLNode *head = getNewLLNode(hashNode);
            fillHashTable(hashTable, hash, head);

            startIdx = -1;
            continue;
        }

        LLNode *matchingNode = findMatchingNode(head, content1+startIdx);
        if(matchingNode) {
            HNode *hashNode = (HNode *) matchingNode->data;
            hashNode->count++;
            startIdx = -1;
            continue;
        }

        HNode *hashNode = getHashNode(content1+startIdx, currentSrc, 1, nodes, &used);
        LLNode *nodeToInsert = getNewLLNode(hashNode);
        fillHashTable(hashTable, hash, nodeToInsert);

        startIdx = -1;
        continue;
    }

    i = 0; startIdx=-1;
    currentSrc = 2;
    for(;i<letterCount2; i++) {
        if(content2[i] != ' ') {
            startIdx = (startIdx == -1) ? i : startIdx;
            continue;
        }

        if(startIdx == -1) {
            continue;
        }

        unsigned char hash = getHash(content2+startIdx, i-startIdx);
        LLNode *head = hashTable[hash];

        if(head == NULL) {
            HNode *hashNode = getHashNode(content1+startIdx, currentSrc, 1, nodes, &used);
            LLNode *head = getNewLLNode(hashNode);
            fillHashTable(hashTable, hash, head);

            startIdx = -1;
            continue;
        }

        LLNode *matchingNode = findMatchingNode(head, content1+startIdx);
        if(matchingNode) {
            HNode *hashNode = (HNode *) matchingNode->data;
            if(hashNode->source != currentSrc) {
                hashNode->count--;
            } else {
                hashNode->count++;
            }
            startIdx = -1;
            continue;
        }

        HNode *hashNode = getHashNode(content1+startIdx, currentSrc, 1, nodes, &used);
        LLNode *nodeToInsert = getNewLLNode(hashNode);
        fillHashTable(hashTable, hash, nodeToInsert);

        startIdx = -1;
        continue;
    }

    return getScoreFromHashTable(hashTable);
}

extern "C"
__global__ void sq(FileContent   fileContent, 
                   BigBoy        fileSize, 
                   BigBoy        n,
                   BigBoy        chunkSize,
                   double*       dScores,
                   uint8_t*      dSyncBuffer)
{
    dSyncBuffer[blockIdx.x] = 0;
    __syncthreads();

    int myIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int myLimit = myIdx + chunkSize;

    int nc2 = findNC2(n);
    if(nc2 <= myIdx) return;

    preprocess(fileContent, myIdx, myLimit);
    __syncthreads();

    FileContent myContent = fileContent + myIdx;
    *myContent = '0' + myIdx/10;
    

    uint32_t firstChunk, secondChunk, i; 
    for(i = 1; i <= (n-1); i++) {
        if(myIdx < (i*n - (i*(i+1))/2)) {
            firstChunk = i;
            break;
        }
    } 

    secondChunk = 1 + myIdx - ((i-1)*n - ((i-1)*i)/2);
    __syncthreads();

    double score = getScore(fileContent+(firstChunk-1)*chunkSize, chunkSize, fileContent+(secondChunk-1)*chunkSize, chunkSize);
    dScores[myIdx] = score;

    __syncthreads();
    dSyncBuffer[blockIdx.x] = 1;

    unsigned int noOfBlocks = (int) ceil(((double)nc2/(double)threadsPerBlock));
    for(i = 0; i < noOfBlocks; i++) {
        if(dSyncBuffer[i] == 0) i--;
    }

    if(myIdx >= n) return;
    uint32_t start = (myIdx*n - (myIdx*(myIdx+1))/2);
    uint32_t end   = ((myIdx+1)*n - ((myIdx+1)*(myIdx+2))/2)
    uint32_t max = dScores[start], maxIndex = start;
    for(i = start; i < end; i++) {
       if(dScores[i] > max) {
            max = dScores[i];
            maxIndex = i;
       } 
    }
    
    dScores[start] = maxIndex;
}

extern "C"
size_t getFilesize(FileName filename) {
    struct stat st;
    if(stat((const char *)filename, &st) != 0) {
        return 0;
    }
    return st.st_size;   
}

extern "C"
void getFileContent(FileName filename, FileContent buffer) {
FILE *file;
size_t nread;

file = fopen((const char *)filename, "r");
assert(file != NULL);

BigBoy filesize = getFilesize(filename);

nread = fread(buffer, 1, filesize, file);
assert(nread == filesize);
assert(ferror(file) == 0);

fclose(file);
}

#ifndef checkCudaErrors
#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)

// These are the inline versions for all of the SDK helper functions
inline void __checkCudaErrors(int err, const char *file, const int line)
{
    if (0 != err)
    {
        fprintf(stderr, "checkCudaErrors() Driver API error = %04d  from file <%s>, line %i.\n",
                err, file, line);
        exit(EXIT_FAILURE);
    }
}
#endif

extern "C"
int main() {

    filename = (unsigned char *) "/home/anand/Desktop/hemanth/phase2/text8";

    FileContent deviceFileBuffer, hostFileBuffer, reorderedFileBuffer;
    BigBoy filesize = getFilesize(filename);
    BigBoy chunkSize = 1024*1024;
    ChunkOrder *deviceReorderInfo, *hostReorderInfo;
    double *dScores, *hScores;
    uint8_t *dSyncBuffer;
    BigBoy n = (int) ceil(((double)filesize/(double)chunkSize));
    BigBoy nc2 = findNC2(n);
    unsigned int threadsPerBlock = 1024;
    unsigned int noOfBlocks = (int) ceil(((double)nc2/(double)threadsPerBlock));

    hostFileBuffer = (FileContent) malloc(filesize+1);
    reorderedFileBuffer = (FileContent) malloc(filesize+1);
    getFileContent(filename, hostFileBuffer);

    checkCudaErrors(hipMalloc((void **)&deviceFileBuffer, filesize));
    checkCudaErrors(hipMalloc((void **)&dScores, sizeof(double)*nc2));
    checkCudaErrors(hipMalloc((void **)&dSyncBuffer, sizeof(uint8_t)*10));
    hostReorderInfo = (ChunkOrder *) malloc(sizeof(ChunkOrder)*n);


    checkCudaErrors(hipMemcpy(deviceFileBuffer, hostFileBuffer, filesize, hipMemcpyHostToDevice));

    printf("Launching CUDA kernal for file size = %llu; chunk size = %llu; no of chunks = %llu;\n", filesize, chunkSize, n);
    printf("                          threadsPerBlock = %u; noOfBlocks = %u; \n", threadsPerBlock, noOfBlocks);

    sq<<<noOfBlocks, threadsPerBlock>>>(deviceFileBuffer, filesize, n, chunkSize, dScores, dSyncBuffer);

    printf("CUDA kernel execution over !!!\n");

    hScores = (double *) malloc(sizeof(double)*nc2);

    checkCudaErrors(hipMemcpy(hostFileBuffer, deviceFileBuffer, filesize, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(hScores, dScores, sizeof(double)*nc2, hipMemcpyDeviceToHost));

    memcpy(reorderedFileBuffer, hostFileBuffer, chunkSize);
    for(uint32_t i = 0; i < n; i++) {
        uint32_t start = (i*n - (i*(i+1))/2);
        uint32_t bestMatchChunkIndex = i + 
        memcpy(reorderedFileBuffer + (i+1)*chunkSize, hostFileBuffer+
    }

    checkCudaErrors(hipFree(deviceFileBuffer));
    return 0;
}
